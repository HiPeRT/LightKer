#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

#include <stdio.h>

#include "core/lk_time.h"
#include "core/lk_utils.h"
#define L_MAX_LENGTH 20

#include "work.cu"

__global__ void kernel(char *str[], unsigned int num[])
{
  printf("[EXAMPLE] block %d will work on data '%s'\n", blockIdx.x, str);
  WORK((const char *) str + blockIdx.x * L_MAX_LENGTH, &num[blockIdx.x]);
  printf("[EXAMPLE] block %d returns %d\n", blockIdx.x, num[blockIdx.x]);  
}

int main()
{
  int numBlocks = 1, numThreads = 1;
  char *h_string, ** d_string;
  unsigned int *h_num, *d_num;
  
  /* Input string */
  printf("Alloc input data (Host)\n");
  checkCudaErrors(hipHostAlloc((void **)&h_string, numBlocks * L_MAX_LENGTH, hipHostMallocDefault));
  printf("Alloc input data (Device)\n");
  checkCudaErrors(hipMalloc((void **)&d_string, numBlocks * L_MAX_LENGTH));
  
  /* Output integers */
  printf("Alloc output data (Host)\n");
  checkCudaErrors(hipHostAlloc((void **)&h_num, numBlocks * sizeof(unsigned int), hipHostMallocDefault));
  printf("Alloc output data (Device)\n");
  checkCudaErrors(hipMalloc((void **)&d_num, numBlocks * sizeof(unsigned int)));
  
  printf("Init data\n");
  /* Init app data */
  char * ptr = h_string;
  for(int i=0; i<numBlocks; i++, ptr += L_MAX_LENGTH*i)
  {
    printf("[EXAMPLE] Invoking INIT_DATA h_string @0x%x\n", _mycast_ ptr);
    INIT_DATA(ptr, i);
  }
  
  printf("Copy data to device\n");
  /* Move data to device. We do the very same way as LK, block by block */
  for(int i=0; i<numBlocks; i++)
    checkCudaErrors(hipMemcpy(d_string + L_MAX_LENGTH*i, h_string + L_MAX_LENGTH*i, L_MAX_LENGTH, hipMemcpyHostToDevice));
    
  printf("Invoke CUDA kernel..\n");
  kernel<<<numBlocks, numThreads>>>(d_string, d_num);
  printf("Wait for CUDA kernel..\n");
  hipDeviceSynchronize();
  
  printf("Copy data from device\n");
  /* Move data to device. We do the very same way as LK, block by block */
  for(int i=0; i<numBlocks; i++)
    checkCudaErrors(hipMemcpy(&h_num[i], &d_num[i], sizeof(unsigned int), hipMemcpyDeviceToHost));
  
  for(int i=0; i<numBlocks; i++)
    CHECK_RESULTS((const char *) h_string + L_MAX_LENGTH*i, h_num[i], numThreads, i);
    
  printf("Dispose data\n");
  checkCudaErrors(hipFree(d_num));
  checkCudaErrors(hipHostFree(h_string));
  checkCudaErrors(hipFree(d_string));
  checkCudaErrors(hipHostFree(h_num));
  
  checkCudaErrors(hipGetLastError());
  
  return 0;
} // main