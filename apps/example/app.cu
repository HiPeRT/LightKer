#include "hip/hip_runtime.h"
#include "../../head/head.h"
#include "../../head/utils.h"

void init_data(data_t **data, int numblocks)
{
	checkCudaErrors(hipHostAlloc((void **)data, numblocks * sizeof(data_t), hipHostMallocDefault));
}

int retrieve_data(data_t *data, int *lk_results, int sm,
                  hipStream_t *backbone_stream)
{
#if 0
    do {
        checkCudaErrors(hipMemcpyAsync(&trig[sm], &d_trig[sm], sizeof(trig_t),
                hipMemcpyDeviceToHost, *backbone_stream));
        log("waiting (retrieve) for %d [%d]\n",  _vcast(trig[sm].to_device), _vcast(trig[sm].from_device));
    } while (_vcast(trig[sm].from_device) != THREAD_FINISHED);

    _vcast(trig[sm].to_device) = THREAD_NOP;
    checkCudaErrors(hipMemcpyAsync(&d_trig[sm], &trig[sm], sizeof(trig_t),
            hipMemcpyHostToDevice, *backbone_stream));
    log("retrieve %d %d\n", _vcast(trig[sm].from_device), _vcast(trig[sm].to_device));
#endif
    return _vcast(lk_results[sm]);
}

int assign_data(data_t *data, int sm, hipStream_t *backbone_stream)
{
	strncpy(data[sm].str, "prova", L_MAX_LENGTH);
	log("assigned data \"%s\" to thread %d\n", (char *)payload, sm);

	return 0;
}

__device__ int work_nocuda(volatile data_t data)
{
	log("Hi! I'm block %d and I'm working on data ''%s'' [NOCUDA]\n", blockIdx.x, data.str);
	clock_t clock_count = 200000;
	clock_t start_clock = clock();
	clock_t clock_offset = 0;
	while (clock_offset < clock_count)
		clock_offset = clock() - start_clock;
	return 1;
}

__device__ int work_cuda(volatile data_t data)
{
	log("Hi! I'm block %d and I'm working on data ''%s'' [CUDA]\n", blockIdx.x, data.str);
	clock_t clock_count = 200000;
	clock_t start_clock = clock();
	clock_t clock_offset = 0;
	if (threadIdx.x == 0) {
		while (clock_offset < clock_count)
			clock_offset = clock() - start_clock;
	}
	return 1;
}
