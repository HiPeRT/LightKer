#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <ctype.h>
#include "header.h"

/*	Inizializza la struttura dati	*/
void initialize (syncon_t *s);

/*	Inizializza il contatore	*/
void init (int *curr_i);

/*	Dalla riga estrae padre e figlio e 
	li inserisce nella struttura dati
*/
void workOnRow (char * row, syncon_t *s, int tab, int *curr_i);

/*	Stampa la struttura dati	*/
void print(syncon_t *s, int n);

/*	Conta il numero di padri di ogni syncon	*/
void contDadsAndSons (syncon_t *s);

/*	Legge una tabella dei link	*/
void readTable ( syncon_t *s, int *curr_i);

/*	Legge dal file passato (il file di test) NBLOCKS istanze di test 
	e inserisce i dati nelle apposite strutture dati
	Ritorna 1 quando il file è terminato, 0 altrimenti
*/
int readNewTest(FILE *infile,int *n_dads,int *syncon,int *dads);

/*	Stampa i risultati di un'esecuzione del kernel	*/
void printResults (int *result, int j);

/*
    Kernel cuda: implementazione della primitiva isKindOf su GPU
    Vengono usati NBLOCKS blocchi, ognuno con NTHREADS thread.
    Ogni blocco riceve un'istanza differente di test che risolve il parallelo.
    
    Dato un array di padri e un synconid la funzione deve trovare un 
    legame tra il syncon e uno dei padri nella gerarchia di iperinomia 
    (memorizzata nella struttura dati s). 
    Se non esiste relazione ritorna -1, se esiste ritorna la profondità
    
*/
__device__ void isKindOf(syncon_t *s, int *synconid, int *n_dads,int *dads,int *result);


/************************************* MAIN ***********************************/
static int curr_i[NSYNCON]; 
syncon_t *s;
int *n_dads, *dads,*syncon,*result;
static FILE *infile = NULL;
struct timespec spec_start, spec_stop;

syncon_t *temp_s;
int *d_result;

static int APP_num_blocks;

static int TEST_IDX = 0;
const int NUM_TESTS = 2;
//const int NUM_TESTS = 300;
static int **g_n_dads, **g_dads, **g_syncon, **g_results;

void lkInitAppData(data_t **data, int numblocks)
{
  log("numblocks is %d NUM_TESTS is %d\n", numblocks, NUM_TESTS);
  totSize =0; 
  totSize += sizeof(syncon_t)*NSYNCON;
  data_t *data_p;

  //PARTE HOST

  checkCudaErrors(hipHostAlloc((void **)&s, NSYNCON*sizeof(syncon_t), hipHostMallocDefault));
  checkCudaErrors(hipHostAlloc((void**)&dads, numblocks*MAXDADS*sizeof(int),hipHostMallocDefault));
  checkCudaErrors(hipHostAlloc((void **)&result, numblocks*sizeof(int), hipHostMallocDefault));
  checkCudaErrors(hipHostAlloc((void**)&syncon,numblocks*sizeof(int),hipHostMallocDefault));
  checkCudaErrors(hipHostAlloc((void**)&n_dads,numblocks*sizeof(int),hipHostMallocDefault));

  initialize(s);
  init(curr_i);
  contDadsAndSons (s);
  readTable(s,curr_i);

  //PARTE DEVICE

  checkCudaErrors(hipHostAlloc((void **)&temp_s, NSYNCON*sizeof(syncon_t), hipHostMallocDefault));
  for ( int i = 0; i < NSYNCON; i++ )
  {
      rel_t * temp;
      checkCudaErrors(hipMalloc( (void**) &temp, s[i].n_rel*sizeof(rel_t) ));
      checkCudaErrors(hipMemcpy(temp, s[i].rel, s[i].n_rel*sizeof(rel_t) , hipMemcpyHostToDevice));
  
      temp_s[i].n_rel = s[i].n_rel;
      temp_s[i].rel = temp;
  }

  checkCudaErrors(hipHostAlloc((void **)data, sizeof(data_t), hipHostMallocDefault));
  data_p = *data;

  checkCudaErrors(hipMalloc((void**)&(data_p->dads), numblocks*MAXDADS*sizeof(int)));
  checkCudaErrors(hipMalloc((void**)&(data_p->synconid), numblocks*sizeof(int)));
  checkCudaErrors(hipMalloc((void**)&(data_p->n_dads), numblocks*sizeof(int)));
  checkCudaErrors(hipHostAlloc((void**)&(data_p->result), numblocks*sizeof(int), hipHostMallocDefault));

  checkCudaErrors(hipMalloc ((void **)&(data_p->syncon), NSYNCON*sizeof(syncon_t)));
  /* Big offload */
  checkCudaErrors(hipMemcpy(data_p->syncon, temp_s, sizeof(syncon_t)*NSYNCON, hipMemcpyHostToDevice));
#if 0
  checkCudaErrors(hipHostAlloc((void **)&(data_p->syncon), NSYNCON*sizeof(syncon_t), hipHostMallocDefault));
  memcpy(data_p->syncon, temp_s, sizeof(syncon_t)*NSYNCON);
#endif

  APP_num_blocks = numblocks;

  int i;

  g_n_dads = (int **)malloc(NUM_TESTS * sizeof(int *));
  g_dads = (int **)malloc(NUM_TESTS * sizeof(int *));
  g_syncon = (int **)malloc(NUM_TESTS * sizeof(int *));
  g_results = (int **) malloc(NUM_TESTS * sizeof(int *));
  
  for (i = 0 ; i < NUM_TESTS ; i++)
  {
      g_n_dads[i] = (int *)malloc(numblocks * sizeof(int));
      g_dads[i] = (int *)malloc(numblocks * MAXDADS * sizeof(int));
      g_syncon[i] = (int *)malloc(numblocks * sizeof(int));
      g_results[i] = (int *) malloc(numblocks * sizeof(int));
  }

  infile = fopen(test, "r");
  if(infile == NULL)
  {
      log("Errore in apertura del file '%s'\n", test);;
      /*perror*/printf("Errore in apertura del file '%s'\n", test);
      exit(2);
  }	

  for (i = 0 ; i < NUM_TESTS ; i++)
  {
    for(int j=0; j<APP_num_blocks; j++)
        g_results[i][j]= 2048; // FAKE VALUE
        
    if (readNewTest(infile, g_n_dads[i], g_syncon[i], g_dads[i]))
    {
      break;
      //return;
    }
  }
  
  for (i = 0 ; i < NUM_TESTS ; i++)
    for(int j=0; j<APP_num_blocks; j++)
      log("Test %d, SM #%d will work on syncon %d\n", i, j, g_syncon[i][j]);
  
  log("done.\n");
}

/* Formerly known as 'retrieve_data' */
int lkRetrieveData(data_t *data, int sm, hipStream_t *backbone_stream)
{
  log("SM #%d\n", sm);
  
  checkCudaErrors(hipMemcpyAsync((int *) &g_results[TEST_IDX-1][sm], (int *) &data->result[sm],
                                  sizeof(int), hipMemcpyDeviceToHost, *backbone_stream));

  log("SM #%d returned (APP) result %d\n", sm, g_results[TEST_IDX-1][sm]);
  
  return 0;
}

/* Formerly known as 'assign_data'*/
int lkSmallOffloadMultiple(data_t *data, dim3 blknum,  hipStream_t *backbone_stream)
{
  checkCudaErrors(hipMemcpyAsync(data->n_dads, g_n_dads[TEST_IDX], APP_num_blocks * sizeof(int), hipMemcpyHostToDevice, *backbone_stream));
  checkCudaErrors(hipMemcpyAsync(data->dads, g_dads[TEST_IDX], APP_num_blocks * MAXDADS * sizeof(int), hipMemcpyHostToDevice, *backbone_stream));
  checkCudaErrors(hipMemcpyAsync(data->synconid, g_syncon[TEST_IDX], APP_num_blocks * sizeof(int), hipMemcpyHostToDevice, *backbone_stream));
  TEST_IDX++;

  return (TEST_IDX < NUM_TESTS);
}

/* For a single SM */
int lkSmallOffload(data_t *data, int sm, hipStream_t *backbone_stream)
{
  checkCudaErrors(hipMemcpyAsync(&data->n_dads[sm], &g_n_dads[TEST_IDX][sm], sizeof(int), hipMemcpyHostToDevice, *backbone_stream));
  checkCudaErrors(hipMemcpyAsync(&data->dads[sm], &g_dads[TEST_IDX][sm], MAXDADS * sizeof(int), hipMemcpyHostToDevice, *backbone_stream));
  checkCudaErrors(hipMemcpyAsync(&data->synconid[sm], &g_syncon[TEST_IDX][sm], sizeof(int), hipMemcpyHostToDevice, *backbone_stream));

  return 0;
}

__device__ int lkWorkCuda(volatile data_t data)
{
    
  isKindOf((syncon_t *)data.syncon, (int *)data.synconid, (int *)data.n_dads, (int *)data.dads, (int *)data.result);

  log("[SM#%d] result is %d\n", blockIdx.x, data.result[blockIdx.x]);
  return LK_EXEC_OK;
}

__device__ int lkWorkNoCuda(volatile data_t data)
{
  isKindOf((syncon_t *)data.syncon, (int *)data.synconid, (int *)data.n_dads, (int *)data.dads, (int *)data.result);

  //log("[SM#%d] result is %d\n", blockIdx.x, data.result[blockIdx.x]);
  return LK_EXEC_OK;
}

/*********************************** FUNZIONI APP-SPECIFIC **********************************/

void initialize (syncon_t *s)
{
  for(int i=0;i<NSYNCON; i++)
  {
      s[i].rel = NULL;
      s[i].n_rel = 0;
  }
}

void init (int *curr_i)
{
  for(int i=0; i<NSYNCON; i++)
      curr_i[i]=0;
}

void workOnRow (char * row, syncon_t *s, int tab, int *curr_i)
{
  int dad;
  int son;
  sscanf (row,"#%d\t#%d",&son,&dad);	

  if (s[son].rel == NULL)
  {
      checkCudaErrors(hipHostAlloc((void **)&s[son].rel, s[son].n_rel*sizeof(rel_t), hipHostMallocPortable));
      totSize+=sizeof(rel_t)*s[son].n_rel;
  }
  s[son].rel[curr_i[son]].tab =tab;
  s[son].rel[curr_i[son]].synconid= dad;
  curr_i[son]++;
}

void print(syncon_t *s, int n)
{
  for(int i=0; i<n; i++)
  {
      printf("Synconid %d\n",i);
      printf("\tPadri: \n");
      for(int j=0;j<s[i].n_rel;j++)
          printf("\tSynconid : %d, tabella %d\n",s[i].rel[j].synconid,s[i].rel[j].tab);
      printf("\tNumero di padri: %d \n",s[i].n_rel);
  }
}

void contDadsAndSons (syncon_t *s)
{
    FILE *infile;
    char row[MAXLEN];
    char *check;
    int dad,son;

    infile=fopen(link0, "r");
    if( infile==NULL ) 
    {
      log("Errore in apertura del file '%s'\n", link0);
      perror("Errore in apertura del file");
      exit(1);
    }

    while(1) 
    {
      check=fgets(row, MAXLEN, infile);

      if( check == NULL )
          break;
      if(row[0]=='/' || row[0]=='\n')
          continue;

      sscanf (row,"#%d\t#%d",&son,&dad);	

      s[son].n_rel++;
    }
    fclose(infile);
}

void readTable ( syncon_t *s, int *curr_i)
{
    FILE *infile;
    char row[MAXLEN];
    char *check;
    
    infile=fopen(link0, "r");
    if( infile==NULL ) 
    {
      perror("Errore in apertura del file");
      exit(1);
    }

    while(1) 
    {
      check=fgets(row, MAXLEN, infile);

      if( check == NULL )
          break;
      if(row[0]=='/' || row[0]=='\n')
          continue;
      workOnRow(row, s, 0,curr_i);
    }

    fclose(infile);
}

int readNewTest(FILE *infile,int *n_dads,int *syncon,int *dads)
{
  char row [MAXLEN];
  char *p,*check;
  bool testErr = false;

  assert(infile);

  //log("FILE %p\n", infile);

  for(int i=0; i<APP_num_blocks; i++)
  {
    check=fgets(row, MAXLEN, infile);
    if (check == NULL)
        return 1;
    p = row;
    n_dads[i] = -1;
    while (*p)
  {
//             printf("p is %c\n", *p);
      if (isdigit(*p))
      { 
        if (n_dads[i] == -1)
        {
          syncon[i] = strtol(p, &p, 10);
          log("syncon[%d] = %d\n", i, syncon[i]);
          if(syncon[i]> NSYNCON-1)
          {
            printf("Errore nel testcase syncon!\n");
            testErr = true;
            break;
          }
                
        }
        else
        {
          dads[i*MAXDADS+n_dads[i]] = strtol(p, &p, 10);
// 					log("dads[%d] = %d\n", i*MAXDADS+n_dads[i], dads[i*MAXDADS+n_dads[i]]);
        }

        n_dads[i]++;
// 				log("n_dads[%d] = %d\n", i, n_dads[i]);
      }
      else 
        p++;
    }
    if(testErr)
    {
      i--;
      testErr = false;
      continue;
    }
  }
  return 0;
}

void printResults (int *result, int j)
{
  for(int i=0; i< APP_num_blocks; i++)	
  {
    if(result[i] == -1)
      printf("Non ci sono relazioni tra i syncon\n");
    else
      printf("La profondità è : %d e sono all'iterazione %d\n",result[i],i+j*APP_num_blocks);
  }
}

__device__ void isKindOf(syncon_t *s, int *synconid, int *n_dads, int *dads, int *_result)
{
  int tid = threadIdx.x;
  int bid = blockIdx.x;
  int threadRunning = blockDim.x;
  int result[4];
      
  __shared__ volatile int done,curr_syn_glob,level;
  __shared__ int s_dim[MAXLEVEL];
  __shared__ rel_t *s_ptr[MAXLEVEL];
  int curr_syn;

  printf("[SM #%d] my tid is %d\n", bid, tid);
  
  return;
  if(tid==0)
  {
    result[bid] = -1;
    level = 0;
    done = 0;
    s_dim[0] = 0;
    s_ptr[0] = NULL;
  }
  
  curr_syn = synconid[bid];
  __syncthreads();

  dbgsrc("Sono prima del while tid %d bid %d threadRunning %d\n", tid, bid, threadRunning);
  
  while (1)
  {
    if(tid == 0)
        dbgsrc("\n\n\n\nNUOVO GIRO:\tControllo il syncon %d\n", curr_syn);

    for(int i=0; i<(n_dads[bid]/threadRunning+1); i++,tid+=threadRunning)
    {
      //log("%d\n", n_dads[bid]/threadRunning+1);
      if(tid < n_dads[bid])
      {
        dbgsrc("Controllo il padre dads[%d] %d, curr %d\n",bid*MAXDADS+tid, dads[bid*MAXDADS+tid], curr_syn);
        if(curr_syn == dads[bid*MAXDADS+tid])
        {
          if(result[bid] == -1) 
            result[bid] = level+1;
          else if (result[bid]> level +1) 
            result[bid] = level+1;
//                   dbgsrc("Il padre fa match a livello  %d. result ora e' %d\n",dads[bid*MAXDADS+tid],level+1,result[bid]);
          dbgsrc("[syncon %d] Trovato un match con %d livello %d. result ora e' %d\n",curr_syn, dads[bid*MAXDADS+tid],level+1,result);
        }
        //dbgsrc("Controllo il padre dads[%d] %d, curr %d\n",bid*MAXDADS+tid, dads[bid*MAXDADS+tid], curr_syn);
      }
    }

    tid = threadIdx.x;
    __syncthreads();
    return;

    if(tid == 0)
    {
      log("Il syncon non ha dato match\n"
          "Numero di padri di %d : %d\n",curr_syn,s[curr_syn].n_rel);

      if(s[curr_syn].n_rel != 0)
      {
        log("Il livello è %d\n",level);

        s_ptr[level] = s[curr_syn].rel;
        s_dim[level] = s[curr_syn].n_rel;
            
        curr_syn = s_ptr[level]->synconid;	//mi sposto sul figlio		
        level++;

        log("curr_ptr punta a %d\t curr_dim è %d\til livello è %d\n",
                s_ptr[level-1]->synconid, s_dim[level-1], level);
        log("Il syncon corrente è %d\n", curr_syn);
      }
      else 
      {
        s_dim[level] = 0;
        while (s_dim[level] < 2 && level >=0)
        {
          log("Sono entrato nel while:\t s_dim vale %d\til livello è %d\n",s_dim[level],level);

          s_ptr[level] = NULL;
          s_dim[level] = 0;
          level--;
          if(level >= 0)
              log("Sono alla fine del while:\t curr_ptr punta a %d\t curr_dim è %d\t il livello è %d\n",
                  s_ptr[level]->synconid, s_dim[level], level);
        }

        if(s_ptr[0] == NULL )
        {
          done = 1;
        }
        else 
        {
          s_ptr[level]++;
          s_dim[level]--;
          curr_syn = s_ptr[level]->synconid; //mi sposto sul fratello
          level++;

          log("Il livello è %d\t il fratello è %d\t e la dimensione è %d\n", level, curr_syn,s_dim[level-1]);
        }
      }
  
      curr_syn_glob = curr_syn;
    }
    __syncthreads();
    if(done)
      break;
    curr_syn = curr_syn_glob;
    __syncthreads();
  } // while

// 	log("Il risultato è %d\n", result[bid]);
} // isKindOf


