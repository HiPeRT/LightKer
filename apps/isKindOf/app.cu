#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <ctype.h>
#include "header.h"

/*	Inizializza la struttura dati	*/
void initialize (syncon_t *s);

/*	Inizializza il contatore	*/
void init (int *curr_i);

/*	Dalla riga estrae padre e figlio e 
	li inserisce nella struttura dati
*/
void workOnRow (char * row, syncon_t *s, int tab, int *curr_i);

/*	Stampa la struttura dati	*/
void print(syncon_t *s, int n);

/*	Conta il numero di padri di ogni syncon	*/
void contDadsAndSons (syncon_t *s);

/*	Legge una tabella dei link	*/
void readTable ( syncon_t *s, int *curr_i);

/*	Legge dal file passato (il file di test) NBLOCKS istanze di test 
	e inserisce i dati nelle apposite strutture dati
	Ritorna 1 quando il file è terminato, 0 altrimenti
*/
int readNewTest(FILE *infile,int *n_dads,int *syncon,int *dads);

/*	Stampa i risultati di un'esecuzione del kernel	*/
void printResults (int *result, int j);

/*
	Kernel cuda: implementazione della primitiva isKindOf su GPU
	Vengono usati NBLOCKS blocchi, ognuno con NTHREADS thread.
	Ogni blocco riceve un'istanza differente di test che risolve il parallelo.
	
	Dato un array di padri e un synconid la funzione deve trovare un 
	legame tra il syncon e uno dei padri nella gerarchia di iperinomia 
	(memorizzata nella struttura dati s). 
	Se non esiste relazione ritorna -1, se esiste ritorna la profondità
	
*/
__device__ void isKindOf(syncon_t *s, int *synconid, int *n_dads,int *dads,int *result);


/************************************* MAIN ***********************************/
int curr_i[NSYNCON]; 
syncon_t *s;
int *n_dads, *dads,*syncon,*result;
FILE *infile;
struct timespec spec_start, spec_stop;

syncon_t *d_s, *temp_s;
int *d_dads, *d_n_dads, *d_syncon, *d_result;

static int APP_num_blocks;

void init_data(data_t **data, int numblocks)
{
	totSize =0; 
	totSize += sizeof(syncon_t)*NSYNCON;
	data_t *data_p;

	log("init_data 1\n");

	//PARTE HOST

	checkCudaErrors(hipHostAlloc((void **)&s, NSYNCON*sizeof(syncon_t), hipHostMallocDefault));
	checkCudaErrors(hipHostAlloc((void**)&dads, numblocks*MAXDADS*sizeof(int),hipHostMallocDefault));
	checkCudaErrors(hipHostAlloc((void **)&result, numblocks*sizeof(int), hipHostMallocDefault));
	checkCudaErrors(hipHostAlloc((void**)&syncon,numblocks*sizeof(int),hipHostMallocDefault));
	checkCudaErrors(hipHostAlloc((void**)&n_dads,numblocks*sizeof(int),hipHostMallocDefault));

	log("init_data 2\n");
	initialize(s);
	log("init_data 3\n");
	init(curr_i);
	log("init_data 4\n");
	contDadsAndSons (s);
	log("init_data 5\n");
	readTable(s,curr_i);
	log("init_data 6\n");

	//PARTE DEVICE

	checkCudaErrors(hipHostAlloc((void **)&temp_s, NSYNCON*sizeof(syncon_t), hipHostMallocDefault));
	for ( int i = 0; i < NSYNCON; i++ )
	{
		rel_t * temp;
		checkCudaErrors(hipMalloc( (void**) &temp, s[i].n_rel*sizeof(rel_t) ));
		checkCudaErrors(hipMemcpy(temp, s[i].rel, s[i].n_rel*sizeof(rel_t) , hipMemcpyHostToDevice));
	
		temp_s[i].n_rel = s[i].n_rel;
		temp_s[i].rel = temp;
	}
	log("init_data 7\n");

	checkCudaErrors(hipMalloc ((void **)&d_s, NSYNCON*sizeof(syncon_t)));
	checkCudaErrors(hipMemcpy(d_s, temp_s, sizeof(syncon_t)*NSYNCON, hipMemcpyHostToDevice));
	log("init_data 8\n");

	checkCudaErrors(hipHostAlloc((void **)data, sizeof(data_t), hipHostMallocDefault));
	log("init_data 8x\n");
	data_p = *data;
	checkCudaErrors(hipHostAlloc((void**)&data_p->dads, numblocks*MAXDADS*sizeof(int), hipHostMallocDefault));
	log("init_data 8a\n");
	checkCudaErrors(hipHostAlloc((void**)&data_p->syncon, numblocks*sizeof(int), hipHostMallocDefault));
	log("init_data 8b\n");
	checkCudaErrors(hipHostAlloc((void**)&data_p->dads, numblocks*sizeof(int), hipHostMallocDefault));
	log("init_data 8c\n");
	checkCudaErrors(hipHostAlloc((void**)&data_p->result, numblocks*sizeof(int), hipHostMallocDefault));
	log("init_data 9\n");

	APP_num_blocks = numblocks;

	//printf("La dimensione totale è %f MB\n\n",(float)totSize/1024/1024);
	
	infile=fopen(test, "r");

	if( infile==NULL ) 
	{
		perror("Errore in apertura del file");
		exit(2);
	}
}

void assign_data(data_t *data, void *payload, int sm)
{		
	if(readNewTest(infile, n_dads, syncon, dads))
		return;		

	//clock_gettime(CLOCK_MONOTONIC, &spec_start);	

	log("ASSIGN\n");
#if 0
	hipMemcpy(data->dads, dads, APP_num_blocks*MAXDADS*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(data->syncon, syncon, APP_num_blocks*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(data->n_dads, n_dads, APP_num_blocks*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(data->result, result, APP_num_blocks*sizeof(int), hipMemcpyHostToDevice);
#endif
}

__device__ int work_cuda(volatile data_t data)
{
	int result;

	log("WORK\n");

	isKindOf((syncon_t *)data.syncon, (int *)data.synconid, (int *)data.n_dads, (int *)data.dads, (int *)data.result);

	result = *(data.result);
	return result;
}

__device__ int work_nocuda(volatile data_t data)
{
	return 0;
}

/***********************************FUNZIONI**********************************/

void initialize (syncon_t *s)
{
	for(int i=0;i<NSYNCON; i++)
	{
		s[i].rel = NULL;
		s[i].n_rel = 0;
	}
}

void init (int *curr_i)
{
	for(int i=0; i<NSYNCON; i++)
		curr_i[i]=0;
}

void workOnRow (char * row, syncon_t *s, int tab, int *curr_i)
{
	int dad;
	int son;
	sscanf (row,"#%d\t#%d",&son,&dad);	

	if (s[son].rel == NULL)
	{
		checkCudaErrors(hipHostAlloc((void **)&s[son].rel, s[son].n_rel*sizeof(rel_t), hipHostMallocPortable));
		totSize+=sizeof(rel_t)*s[son].n_rel;
	}
	s[son].rel[curr_i[son]].tab =tab;
	s[son].rel[curr_i[son]].synconid= dad;
	curr_i[son]++;
}

void print(syncon_t *s, int n)
{
	for(int i=0; i<n; i++)
	{
		printf("Synconid %d\n",i);
		printf("\tPadri: \n");
		for(int j=0;j<s[i].n_rel;j++)
			printf("\tSynconid : %d, tabella %d\n",s[i].rel[j].synconid,s[i].rel[j].tab);
		printf("\tNumero di padri: %d \n",s[i].n_rel);
	}
}

void contDadsAndSons (syncon_t *s)
{
	FILE *infile;
	char row[MAXLEN];
	char *check;
	int dad,son;

	infile=fopen(link0, "r");
	if( infile==NULL ) 
	{
		perror("Errore in apertura del file");
		exit(1);
	}	

	while(1) 
	{
		check=fgets(row, MAXLEN, infile);

		if( check == NULL )
			break;
		if(row[0]=='/' || row[0]=='\n')
			continue;

		sscanf (row,"#%d\t#%d",&son,&dad);	

		s[son].n_rel++;
	}
	fclose(infile);
}

void readTable ( syncon_t *s, int *curr_i)
{
	FILE *infile;
	char row[MAXLEN];
	char *check;
	
	infile=fopen(link0, "r");
	if( infile==NULL ) 
	{
		perror("Errore in apertura del file");
		exit(1);
	}	

	while(1) 
	{
		check=fgets(row, MAXLEN, infile);

		if( check == NULL )
			break;
		if(row[0]=='/' || row[0]=='\n')
			continue;
		workOnRow(row, s, 0,curr_i);

	}

	fclose(infile);
}

int readNewTest(FILE *infile,int *n_dads,int *syncon,int *dads)
{
	char row [MAXLEN];
	char *p,*check;
	bool testErr = false;

	for(int i=0; i<APP_num_blocks; i++)
	{
		check=fgets(row, MAXLEN, infile);
		if (check == NULL)
			return 1;
		p = row;
		n_dads[i] = -1;
		while (*p) 
		{
			if (isdigit(*p)) 
			{ 
				if (n_dads[i] == -1)
				{
					syncon[i] = strtol(p, &p, 10);
					if(syncon[i]> NSYNCON-1)
					{
						//printf("Errore nel testcase syncon!\n");
						testErr = true;
						break;
					}
						
				}
				else
				{
					dads[i*MAXDADS+n_dads[i]] = strtol(p, &p, 10);
				}

				n_dads[i]++;
			} 
			else 
			    p++;
		}
		if(testErr)
		{
			i--;
			testErr = false;
			continue;
		}
			
	}
	return 0;
}

void printResults (int *result, int j)
{
	for(int i=0; i< APP_num_blocks; i++)	
	{
		if(result[i] == -1)
			printf("Non ci sono relazioni tra i syncon\n");
		else
			printf("La profondità è : %d e sono all'iterazione %d\n",result[i],i+j*APP_num_blocks);
	}
}

__device__ void isKindOf(syncon_t *s, int *synconid, int *n_dads, int *dads, int *result)
{
	int tid = threadIdx.x;
	int bid = blockIdx.x;
	int threadRunning = blockDim.x;
	
	__shared__ volatile int done,curr_syn_glob,level;
	__shared__ int s_dim[MAXLEVEL];
	__shared__ rel_t *s_ptr[MAXLEVEL];
	
	int curr_syn;

	if(tid==0)
	{
		result[bid] = -1;
		level = 0;
		done = 0;
		s_dim[0] = 0;
		s_ptr[0] = NULL;
	}

	curr_syn =synconid[bid];
	__syncthreads();

	dbgsrc("Sono prima del while\n");
	
	while (1)
	{
		if(tid==0)
			dbgsrc("\n\n\n\nNUOVO GIRO:\tControllo il syncon %d\n", curr_syn);

		for(int i=0; i<(n_dads[bid]/threadRunning+1); i++,tid+=threadRunning)
			if(tid < n_dads[bid])
			{
				dbgsrc("Controllo il padre %d\n",dads[bid*MAXDADS+tid]);
				if(curr_syn == dads[bid*MAXDADS+tid])
				{
					if(result[bid] == -1)
						result[bid]= level+1;
					else if (result[bid]> level +1)
						result[bid] = level+1;
				}
			}

		tid = threadIdx.x;
		__syncthreads();

		if(tid == 0)
		{

			dbgsrc(	"Il syncon non ha dato match\n"
				"Numero di padri di %d : %d\n",curr_syn,s[curr_syn].n_rel);

			if(s[curr_syn].n_rel != 0)
			{	
				dbgsrc("Il livello è %d\n",level);
	
				s_ptr[level] = s[curr_syn].rel;
				s_dim[level] = s[curr_syn].n_rel;
					
				curr_syn = s_ptr[level]->synconid;	//mi sposto sul figlio		
				level++;
	
				dbgsrc(	"curr_ptr punta a %d\t curr_dim è %d\til livello è %d\n",
						s_ptr[level-1]->synconid, s_dim[level-1], level);
				dbgsrc("Il syncon corrente è %d\n", curr_syn);
			}
			else 
			{
				s_dim[level] = 0;
				while (s_dim[level] < 2 && level >=0)
				{
					dbgsrc(	"Sono entrato nel while:\t s_dim vale %d"
							"\til livello è %d\n",s_dim[level],level);
	
					s_ptr[level] = NULL;
					s_dim[level] = 0;
					level--;
					if(level >= 0)
						dbgsrc(	"Sono alla fine del while:\t curr_ptr punta a %d\t "
							"curr_dim è %d\t il livello è %d\n",
							s_ptr[level]->synconid, s_dim[level], level);
				}

				if(s_ptr[0] == NULL )
				{
					done = 1;
				}
				else 
				{	
					s_ptr[level]++;
					s_dim[level]--;
					curr_syn = s_ptr[level]->synconid; //mi sposto sul fratello
					level++;
	
					dbgsrc(	"Il livello è %d\t il fratello è %d\t e la dimensione è %d\n",
						level, curr_syn,s_dim[level-1]);
				}
			}
		
			curr_syn_glob = curr_syn;
		}
		__syncthreads();
		if(done)
			break;
		curr_syn = curr_syn_glob;
		__syncthreads();
		
	}
	
	dbgsrc("Il risultato è %d\n",*result);
}


