#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <ctype.h>
#include "header.h"

/*	Inizializza la struttura dati	*/
void initialize (syncon_t *s);

/*	Inizializza il contatore	*/
void init (int *curr_i);

/*	Dalla riga estrae padre e figlio e 
	li inserisce nella struttura dati
*/
void workOnRow (char * row, syncon_t *s, int tab, int *curr_i);

/*	Stampa la struttura dati	*/
void print(syncon_t *s, int n);

/*	Conta il numero di padri di ogni syncon	*/
void contDadsAndSons (syncon_t *s);

/*	Legge una tabella dei link	*/
void readTable ( syncon_t *s, int *curr_i);

/*	Legge dal file passato (il file di test) NBLOCKS istanze di test 
	e inserisce i dati nelle apposite strutture dati
	Ritorna 1 quando il file è terminato, 0 altrimenti
*/
int readNewTest(FILE *infile,int *n_dads,int *syncon,int *dads);

/*	Stampa i risultati di un'esecuzione del kernel	*/
void printResults (int *result, int j);

/*
	Kernel cuda: implementazione della primitiva isKindOf su GPU
	Vengono usati NBLOCKS blocchi, ognuno con NTHREADS thread.
	Ogni blocco riceve un'istanza differente di test che risolve il parallelo.
	
	Dato un array di padri e un synconid la funzione deve trovare un 
	legame tra il syncon e uno dei padri nella gerarchia di iperinomia 
	(memorizzata nella struttura dati s). 
	Se non esiste relazione ritorna -1, se esiste ritorna la profondità
	
*/
__device__ void isKindOf(syncon_t *s, int *synconid, int *n_dads,int *dads,int *result);


/************************************* MAIN ***********************************/
static int curr_i[NSYNCON]; 
syncon_t *s;
int *n_dads, *dads,*syncon,*result;
static FILE *infile = NULL;
struct timespec spec_start, spec_stop;

syncon_t *temp_s;
int *d_result;

static int APP_num_blocks;

static int TEST_IDX = 0;
const int NUM_TESTS = 1;
static int **g_n_dads, **g_dads, **g_syncon;

void init_data(data_t **data, int numblocks)
{
	totSize =0; 
	totSize += sizeof(syncon_t)*NSYNCON;
	data_t *data_p;

	//PARTE HOST

	checkCudaErrors(hipHostAlloc((void **)&s, NSYNCON*sizeof(syncon_t), hipHostMallocDefault));
	checkCudaErrors(hipHostAlloc((void**)&dads, numblocks*MAXDADS*sizeof(int),hipHostMallocDefault));
	checkCudaErrors(hipHostAlloc((void **)&result, numblocks*sizeof(int), hipHostMallocDefault));
	checkCudaErrors(hipHostAlloc((void**)&syncon,numblocks*sizeof(int),hipHostMallocDefault));
	checkCudaErrors(hipHostAlloc((void**)&n_dads,numblocks*sizeof(int),hipHostMallocDefault));

	initialize(s);
	init(curr_i);
	contDadsAndSons (s);
	readTable(s,curr_i);

	//PARTE DEVICE

	checkCudaErrors(hipHostAlloc((void **)&temp_s, NSYNCON*sizeof(syncon_t), hipHostMallocDefault));
	for ( int i = 0; i < NSYNCON; i++ )
	{
		rel_t * temp;
		checkCudaErrors(hipMalloc( (void**) &temp, s[i].n_rel*sizeof(rel_t) ));
		checkCudaErrors(hipMemcpy(temp, s[i].rel, s[i].n_rel*sizeof(rel_t) , hipMemcpyHostToDevice));
	
		temp_s[i].n_rel = s[i].n_rel;
		temp_s[i].rel = temp;
	}

	checkCudaErrors(hipHostAlloc((void **)data, sizeof(data_t), hipHostMallocDefault));
	data_p = *data;

	checkCudaErrors(hipHostAlloc((void**)&(data_p->dads), numblocks*MAXDADS*sizeof(int), hipHostMallocDefault));
	checkCudaErrors(hipHostAlloc((void**)&(data_p->synconid), numblocks*sizeof(int), hipHostMallocDefault));
	checkCudaErrors(hipHostAlloc((void**)&(data_p->n_dads), numblocks*sizeof(int), hipHostMallocDefault));
	checkCudaErrors(hipHostAlloc((void**)&(data_p->result), numblocks*sizeof(int), hipHostMallocDefault));

	checkCudaErrors(hipMalloc ((void **)&(data_p->syncon), NSYNCON*sizeof(syncon_t)));
	checkCudaErrors(hipMemcpy(data_p->syncon, temp_s, sizeof(syncon_t)*NSYNCON, hipMemcpyHostToDevice));
#if 0
	checkCudaErrors(hipHostAlloc((void **)&(data_p->syncon), NSYNCON*sizeof(syncon_t), hipHostMallocDefault));
	memcpy(data_p->syncon, temp_s, sizeof(syncon_t)*NSYNCON);
#endif

	APP_num_blocks = numblocks;

	int i;

	g_n_dads = (int **)malloc(NUM_TESTS * sizeof(int *));
	g_dads = (int **)malloc(NUM_TESTS * sizeof(int *));
	g_syncon = (int **)malloc(NUM_TESTS * sizeof(int *));
	for (i = 0 ; i < NUM_TESTS ; i++) {
		g_n_dads[i] = (int *)malloc(numblocks * sizeof(int));
		g_dads[i] = (int *)malloc(numblocks * MAXDADS * sizeof(int));
		g_syncon[i] = (int *)malloc(numblocks * sizeof(int));
	}

	infile=fopen(test, "r");
	if( infile==NULL ) {
		perror("Errore in apertura del file");
		exit(2);
	}	

	for (i = 0 ; i < NUM_TESTS ; i++) {
		if (readNewTest(infile, g_n_dads[i], g_syncon[i], g_dads[i]))
			return;
	}
}

int assign_data(data_t *data, int sm)
{
	memcpy(data->n_dads, g_n_dads[TEST_IDX], APP_num_blocks * sizeof(int));
	memcpy(data->dads, g_dads[TEST_IDX], APP_num_blocks * MAXDADS * sizeof(int));
	memcpy(data->synconid, g_syncon[TEST_IDX], APP_num_blocks * sizeof(int));
	TEST_IDX++;

	return (TEST_IDX < NUM_TESTS);
}

__device__ int work_cuda(volatile data_t data)
{
	int result;

	isKindOf((syncon_t *)data.syncon, (int *)data.synconid, (int *)data.n_dads, (int *)data.dads, (int *)data.result);

	result = *(data.result);
	return result;
}

__device__ int work_nocuda(volatile data_t data)
{
	return 0;
}

/***********************************FUNZIONI**********************************/

void initialize (syncon_t *s)
{
	for(int i=0;i<NSYNCON; i++)
	{
		s[i].rel = NULL;
		s[i].n_rel = 0;
	}
}

void init (int *curr_i)
{
	for(int i=0; i<NSYNCON; i++)
		curr_i[i]=0;
}

void workOnRow (char * row, syncon_t *s, int tab, int *curr_i)
{
	int dad;
	int son;
	sscanf (row,"#%d\t#%d",&son,&dad);	

	if (s[son].rel == NULL)
	{
		checkCudaErrors(hipHostAlloc((void **)&s[son].rel, s[son].n_rel*sizeof(rel_t), hipHostMallocPortable));
		totSize+=sizeof(rel_t)*s[son].n_rel;
	}
	s[son].rel[curr_i[son]].tab =tab;
	s[son].rel[curr_i[son]].synconid= dad;
	curr_i[son]++;
}

void print(syncon_t *s, int n)
{
	for(int i=0; i<n; i++)
	{
		printf("Synconid %d\n",i);
		printf("\tPadri: \n");
		for(int j=0;j<s[i].n_rel;j++)
			printf("\tSynconid : %d, tabella %d\n",s[i].rel[j].synconid,s[i].rel[j].tab);
		printf("\tNumero di padri: %d \n",s[i].n_rel);
	}
}

void contDadsAndSons (syncon_t *s)
{
	FILE *infile;
	char row[MAXLEN];
	char *check;
	int dad,son;

	infile=fopen(link0, "r");
	if( infile==NULL ) 
	{
		perror("Errore in apertura del file");
		exit(1);
	}	

	while(1) 
	{
		check=fgets(row, MAXLEN, infile);

		if( check == NULL )
			break;
		if(row[0]=='/' || row[0]=='\n')
			continue;

		sscanf (row,"#%d\t#%d",&son,&dad);	

		s[son].n_rel++;
	}
	fclose(infile);
}

void readTable ( syncon_t *s, int *curr_i)
{
	FILE *infile;
	char row[MAXLEN];
	char *check;
	
	infile=fopen(link0, "r");
	if( infile==NULL ) 
	{
		perror("Errore in apertura del file");
		exit(1);
	}	

	while(1) 
	{
		check=fgets(row, MAXLEN, infile);

		if( check == NULL )
			break;
		if(row[0]=='/' || row[0]=='\n')
			continue;
		workOnRow(row, s, 0,curr_i);

	}

	fclose(infile);
}

int readNewTest(FILE *infile,int *n_dads,int *syncon,int *dads)
{
	char row [MAXLEN];
	char *p,*check;
	bool testErr = false;

	assert(infile);

	//log("FILE %p\n", infile);

	for(int i=0; i<APP_num_blocks; i++) {
		check=fgets(row, MAXLEN, infile);
		if (check == NULL)
			return 1;
		p = row;
		n_dads[i] = -1;
		while (*p) {
			if (isdigit(*p)) { 
				if (n_dads[i] == -1) {
					syncon[i] = strtol(p, &p, 10);
					log("syncon[%d] = %d\n", i, syncon[i]);
					if(syncon[i]> NSYNCON-1)
					{
						//printf("Errore nel testcase syncon!\n");
						testErr = true;
						break;
					}
						
				} else {
					dads[i*MAXDADS+n_dads[i]] = strtol(p, &p, 10);
					log("dads[%d] = %d\n", i*MAXDADS+n_dads[i], dads[i*MAXDADS+n_dads[i]]);
				}

				n_dads[i]++;
				log("n_dads[%d] = %d\n", i, n_dads[i]);
			} else 
			    p++;
		}
		if(testErr) {
			i--;
			testErr = false;
			continue;
		}
	}
	return 0;
}

void printResults (int *result, int j)
{
	for(int i=0; i< APP_num_blocks; i++)	
	{
		if(result[i] == -1)
			printf("Non ci sono relazioni tra i syncon\n");
		else
			printf("La profondità è : %d e sono all'iterazione %d\n",result[i],i+j*APP_num_blocks);
	}
}

__device__ void isKindOf(syncon_t *s, int *synconid, int *n_dads, int *dads, int *result)
{
	int tid = threadIdx.x;
	int bid = blockIdx.x;
	int threadRunning = blockDim.x;

	__shared__ volatile int done,curr_syn_glob,level;
	__shared__ int s_dim[MAXLEVEL];
	__shared__ rel_t *s_ptr[MAXLEVEL];
	int curr_syn;

	if(tid==0)
	{
		result[bid] = -1;
		level = 0;
		done = 0;
		s_dim[0] = 0;
		s_ptr[0] = NULL;
	}

	curr_syn =synconid[bid];
	__syncthreads();

	dbgsrc("Sono prima del while %d %d %d\n", tid, bid, threadRunning);

	while (1)
	{
		if(tid==0)
			log("\n\n\n\nNUOVO GIRO:\tControllo il syncon %d\n", curr_syn);

		for(int i=0; i<(n_dads[bid]/threadRunning+1); i++,tid+=threadRunning) {
			log("%d\n", n_dads[bid]/threadRunning+1);
			if(tid < n_dads[bid])
			{
				log("Controllo il padre dads[%d] %d, curr %d\n",bid*MAXDADS+tid, dads[bid*MAXDADS+tid], curr_syn);
				if(curr_syn == dads[bid*MAXDADS+tid])
				{
					if(result[bid] == -1)
						result[bid]= level+1;
					else if (result[bid]> level +1)
						result[bid] = level+1;
				}
			}
		}

		tid = threadIdx.x;
		__syncthreads();


		if(tid == 0)
		{

			log(	"Il syncon non ha dato match\n"
				"Numero di padri di %d : %d\n",curr_syn,s[curr_syn].n_rel);

			if(s[curr_syn].n_rel != 0)
			{	
				log("Il livello è %d\n",level);
	
				s_ptr[level] = s[curr_syn].rel;
				s_dim[level] = s[curr_syn].n_rel;
					
				curr_syn = s_ptr[level]->synconid;	//mi sposto sul figlio		
				level++;
	
				log(	"curr_ptr punta a %d\t curr_dim è %d\til livello è %d\n",
						s_ptr[level-1]->synconid, s_dim[level-1], level);
				log("Il syncon corrente è %d\n", curr_syn);
			}
			else 
			{
				s_dim[level] = 0;
				while (s_dim[level] < 2 && level >=0)
				{
					log(	"Sono entrato nel while:\t s_dim vale %d"
							"\til livello è %d\n",s_dim[level],level);
	
					s_ptr[level] = NULL;
					s_dim[level] = 0;
					level--;
					if(level >= 0)
						log(	"Sono alla fine del while:\t curr_ptr punta a %d\t "
							"curr_dim è %d\t il livello è %d\n",
							s_ptr[level]->synconid, s_dim[level], level);
				}

				if(s_ptr[0] == NULL )
				{
					done = 1;
				}
				else 
				{	
					s_ptr[level]++;
					s_dim[level]--;
					curr_syn = s_ptr[level]->synconid; //mi sposto sul fratello
					level++;
	
					log(	"Il livello è %d\t il fratello è %d\t e la dimensione è %d\n",
						level, curr_syn,s_dim[level-1]);
				}
			}
		
			curr_syn_glob = curr_syn;
		}
		__syncthreads();
		if(done)
			break;
		curr_syn = curr_syn_glob;
		__syncthreads();
	}

	log("Il risultato è %d\n",*result);
}


