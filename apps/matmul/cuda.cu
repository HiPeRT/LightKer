#include "hip/hip_runtime.h"
//Matrix multiplication using shared and non shared kernal
#include <stdio.h>
#include <math.h>

#define TILE_WIDTH 2

#include "core/lk_time.h"

/*matrix multiplication kernels*/
// #include "work.cu"

//non shared
__global__ void
MatrixMul( float *Md , float *Nd , float *Pd , const int WIDTH )
{
  // calculate thread id
  unsigned int col = TILE_WIDTH*blockIdx.x + threadIdx.x ;
  unsigned int row = TILE_WIDTH*blockIdx.y + threadIdx.y ;
  printf("blockIdx %d %d threadIdx %d %d ==> col %hu row %hu Md[0] %f\n", blockIdx.x, blockIdx.y, threadIdx.x, threadIdx.y, col, row, Md[0]);
  Pd[row*WIDTH + col] = 0;
  
//   WORK(Md, Nd, Pd, WIDTH, col, row);
  for (int k = 0 ; k<WIDTH ; k++ )
  {
    Pd[row*WIDTH + col] += Md[row * WIDTH + k ] * Nd[ k * WIDTH + col] ;
//     printf("k %d Md[%d] %f Nd[%d] %f Pd[%d] %f\n", k, row * WIDTH + k, Md[row * WIDTH + k ], k * WIDTH + col, Nd[ k * WIDTH + col], row*WIDTH + col, Pd[row*WIDTH + col]);
  }
  
}

// main routine
int main ()
{
//   #define WIDTH 8
  const int WIDTH = 8;
  float array1_h[WIDTH][WIDTH] ,array2_h[WIDTH][WIDTH],
        result_array_h[WIDTH][WIDTH] ,M_result_array_h[WIDTH][WIDTH]  ;
  float *array1_d , *array2_d ,*result_array_d  ,*M_result_array_d ; // device array
  int i , j ;
  
  struct timespec spec_start, spec_stop, app_start, app_stop;
  long app_total;
  
  //input in host array
  for ( i = 0 ; i<WIDTH ; i++ )
  {
    for (j = 0 ; j<WIDTH ; j++ )
    {
      array1_h[i][j] = 1 ;
      array2_h[i][j] = 2 ;
    }
  }

  //create device array hipMalloc ( (void **)&array_name, sizeofmatrixinbytes) ;
  hipMalloc((void **) &array1_d , WIDTH*WIDTH*sizeof (int) ) ;
  hipMalloc((void **) &array2_d , WIDTH*WIDTH*sizeof (int) ) ;

  //copy host array to device array; hipMemcpy ( dest , source , WIDTH , direction )
  hipMemcpy ( array1_d , array1_h , WIDTH*WIDTH*sizeof (int) , hipMemcpyHostToDevice ) ;
  hipMemcpy ( array2_d , array2_h , WIDTH*WIDTH*sizeof (int) , hipMemcpyHostToDevice ) ;

  //allocating memory for resultent device array
  hipMalloc((void **) &result_array_d , WIDTH*WIDTH*sizeof (int) ) ;
  hipMalloc((void **) &M_result_array_d , WIDTH*WIDTH*sizeof (int) ) ;

  //calling kernal
  dim3 dimGrid ( WIDTH/TILE_WIDTH , WIDTH/TILE_WIDTH ,1 ) ;
  dim3 dimBlock( TILE_WIDTH, TILE_WIDTH, 1 ) ;
  printf("dimGrid %d, %d dimBlock %d %d\n", dimGrid.x, dimGrid.y, dimBlock.x, dimBlock.y);
  
  // Change if 0 to if 1 for running non shared code and make if 0 for shared memory code
#if 1
    clock_gettime(CLOCK_MONOTONIC, &app_start);

    MatrixMul <<<dimGrid,dimBlock>>> ( array1_d, array2_d, M_result_array_d, WIDTH) ;
    clock_gettime(CLOCK_MONOTONIC, &app_stop);
    app_total = clock_getdiff_nsec(app_start, app_stop);
    GETTIME_LOG("[TOTAL] %lu\n", app_total);
#endif

#if 0

    MatrixMulSh<<<dimGrid,dimBlock>>> ( array1_d , array2_d ,M_result_array_d , WIDTH) ;

#endif

  // all gpu function blocked till kernel is working
  //copy back result_array_d to result_array_h

  hipMemcpy(M_result_array_h , M_result_array_d , WIDTH*WIDTH*sizeof(int) , hipMemcpyDeviceToHost) ;

  //printf the result array
  for ( i = 0 ; i<WIDTH ; i++ )
  {
    for ( j = 0 ; j < WIDTH ; j++ )
    {
      printf ("%f   ",M_result_array_h[i][j] ) ;
    }
    printf ("\n") ;
  }

}


// shared
__global__ void
MatrixMulSh( float *Md , float *Nd , float *Pd , const int WIDTH )
{

  //Taking shared array to break the MAtrix in Tile widht and fatch them in that array per ele
  __shared__ float Mds [TILE_WIDTH][TILE_WIDTH] ;
  __shared__ float Nds [TILE_WIDTH][TILE_WIDTH] ;
  // calculate thread id
  unsigned int col = TILE_WIDTH*blockIdx.x + threadIdx.x ;
  unsigned int row = TILE_WIDTH*blockIdx.y + threadIdx.y ;

  for (int m = 0 ; m<WIDTH/TILE_WIDTH ; m++ ) // m indicate number of phase
  {
    Mds[threadIdx.y][threadIdx.x] =  Md[row*WIDTH + (m*TILE_WIDTH + threadIdx.x)]  ;
    Nds[threadIdx.y][threadIdx.x] =  Nd[ ( m*TILE_WIDTH + threadIdx.y) * WIDTH + col] ;
    __syncthreads() ; // for syncronizeing the threads

    // Do for tile
    for ( int k = 0; k<TILE_WIDTH ; k++ )
      Pd[row*WIDTH + col]+= Mds[threadIdx.x][k] * Nds[k][threadIdx.y] ;
    __syncthreads() ; // for syncronizeing the threads

  }
}