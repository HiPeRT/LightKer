#include "lk_utils.h"

void lkDeviceAlloc(void** pDev, size_t size)
{
  checkCudaErrors(hipMalloc((void **) pDev, size));
}

void lkHostAlloc(void **pHost, size_t size)
{
  checkCudaErrors(hipHostAlloc((void **) pHost, size, hipHostMallocDefault));
}

void lkMemcpy(void *dst, const void *src, size_t count, enum hipMemcpyKind kind)
{
  checkCudaErrors(hipMemcpyAsync(dst, src, count, kind, backbone_stream));
  hipStreamSynchronize(backbone_stream);
}

void lkMemcpyToDevice(void *dstDev, const void *srcHost, size_t count)
{
  lkMemcpy(dstDev, srcHost, count, hipMemcpyHostToDevice);
}

void lkMemcpyFromDevice(void *dstHost, const void *srcDev, size_t count)
{
  lkMemcpy(dstHost, srcDev, count, hipMemcpyDeviceToHost);
}
