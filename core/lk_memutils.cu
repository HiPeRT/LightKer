void lkDeviceAlloc(void** dataHostPtr, size_t size)
{
  checkCudaErrors(hipMalloc((void **) dataHostPtr, size));
}

void lkHostAlloc(void **pHost, size_t size)
{
  checkCudaErrors(hipHostAlloc((void **) pHost, size, hipHostMallocDefault));
}

void lkMemcpy(void *dst, const void *src, size_t count, enum hipMemcpyKind kind)
{
  checkCudaErrors(hipMemcpyAsync(dst, src, count, kind, backbone_stream));
  hipStreamSynchronize(backbone_stream);
}

void lkMemcpyToDevice(void *dst, const void *src, size_t count)
{
  lkMemcpy(dst, src, count, hipMemcpyHostToDevice);
}

void lkMemcpyFromDevice(void *dst, const void *src, size_t count, enum hipMemcpyKind kind)
{
  lkMemcpy(dst, src, count, hipMemcpyDeviceToHost);
}
