#include "hip/hip_runtime.h"
#ifndef __MAILBOX_H__
#define __MAILBOX_H__

/* Mailbox types */

typedef int mailbox_elem_t;
typedef mailbox_elem_t mailbox_t[MAX_NUM_BLOCKS];

mailbox_elem_t *d_to_device, *d_from_device, *h_to_device, *h_from_device;

#define lkHToDevice(_sm) _vcast(h_to_device[_sm])
#define lkHFromDevice(_sm) _vcast(h_from_device[_sm])
#define lkDToDevice(_sm) _vcast(d_to_device[_sm])
#define lkDFromDevice(_sm) _vcast(d_from_device[_sm])

#include "lk_utils.h"

  // FIXME handle "my" stream
extern hipStream_t backbone_stream;

int lkMailboxInit(hipStream_t stream = 0)
{
  log("sizeof(mailbox_elem_t) %d sizeof(mailbox_t) %d\n", sizeof(mailbox_elem_t), sizeof(mailbox_t));
  /* hipHostAlloc: shared between host and GPU */
  checkCudaErrors(hipHostAlloc((void **)&h_to_device, sizeof(mailbox_t), hipHostMallocDefault));
  checkCudaErrors(hipMalloc((void **)&d_to_device, sizeof(mailbox_t)));
  checkCudaErrors(hipHostAlloc((void **)&h_from_device, sizeof(mailbox_t), hipHostMallocDefault));
  checkCudaErrors(hipMalloc((void **)&d_from_device, sizeof(mailbox_t)));
  
  log("Created host-side mailbox @0x%x (TO) and 0x%x (FROM). Size is %d.\n",
      _mycast_ &h_to_device[0], _mycast_ &h_from_device[0], sizeof(mailbox_t));
  log("Created device-side mailbox @0x%x (TO) and 0x%x (FROM). Size is %d.\n",
       _mycast_ &d_to_device[0], _mycast_ &d_from_device[0], sizeof(mailbox_t));
  
  return 0;
} // lkMailboxInit

void lkMailboxFree()
{
  checkCudaErrors(hipFree(d_from_device));
  checkCudaErrors(hipHostFree(h_from_device));
  checkCudaErrors(hipFree(d_to_device));
  checkCudaErrors(hipHostFree(h_to_device));
} // lkMailboxFree

ALWAYS_INLINE void
lkMailboxPrint(const char *fn_name, int sm)
{
  log("[%s] to_device %s (%d), from_device %s (%d)\n", fn_name,
      getFlagName(lkHToDevice(sm)), lkHToDevice(sm), getFlagName(lkHFromDevice(sm)), lkHFromDevice(sm));
} // lkMailboxPrint

ALWAYS_INLINE void
lkMailboxSync()
{
  hipStreamSynchronize(backbone_stream);
} // lkMailboxSync

ALWAYS_INLINE void
lkMailboxFlushSM(bool to_device, int sm)
{
//   log("direction: %s, sm %d\n", to_device ? "to_device": "from_device", sm);
  if(to_device)
    checkCudaErrors(hipMemcpyAsync(&d_to_device[0], &h_to_device[0], sizeof(mailbox_elem_t), hipMemcpyHostToDevice, backbone_stream));
  else
    checkCudaErrors(hipMemcpyAsync(&h_from_device[0], &d_from_device[0], sizeof(mailbox_elem_t), hipMemcpyDeviceToHost, backbone_stream));
} // lkMailboxFlushSM

ALWAYS_INLINE void
lkMailboxFlush(bool to_device)
{
//   log("direction: %s\n", to_device ? "to_device": "from_device");
  if(to_device)
    checkCudaErrors(hipMemcpyAsync(&d_to_device[0], &h_to_device[0], sizeof(mailbox_t), hipMemcpyHostToDevice, backbone_stream));
  else
    checkCudaErrors(hipMemcpyAsync(&h_from_device[0], &d_from_device[0], sizeof(mailbox_t), hipMemcpyDeviceToHost, backbone_stream));
} // lkMailboxFlush

#endif /* __MAILBOX_H__ */
