#include "hip/hip_runtime.h"
// #include <stdio.h>
#include <assert.h>
// #include <hip/hip_runtime.h>
// #include <hip/hip_runtime.h>
// #include <hip/hip_runtime_api.h>
// #include <unistd.h>
// #include <unistd.h>
// #include <time.h>
// #include <math.h>
// #include <inttypes.h>
// #include <getopt.h>
// #include <stdlib.h>

/* LK internal headers */
#include "head.h"
#include "utils.h"
/* To treat APP specific data_t type */
#include "data.h"
// #include "app.cu"
// 
// #include "lk_host.h"

/* Low-level functionalities */

/* Initialize the triggers and start the kernel.
 * formerly known as 'init'
 */
void lkLaunch(void (*kernel) (volatile trig_t *, volatile data_t *, int *),
			  trig_t *trig, trig_t *d_trig, data_t *data, int *results,
			  dim3 blkdim, dim3 blknum, int shmem,
			  hipStream_t *stream_kernel, hipStream_t *backbone_stream)
{
	int wg = blknum.x;

	// trigger initialization
	for (int i = 0; i < wg; i++) {
		_vcast(trig[i].to_device) = THREAD_NOP;
	}
	checkCudaErrors(hipMemcpyAsync(d_trig, trig, sizeof(trig_t) * wg,
			hipMemcpyHostToDevice, *backbone_stream));

	kernel <<< blknum, blkdim, shmem, *stream_kernel >>> (d_trig, data, results);
}

/* Order the given sm to start working.
 * Formerly known as 'work'
 */
void lkTriggerSM(trig_t *trig, trig_t *d_trig, int sm, dim3 blknum, hipStream_t *backbone_stream)
{
    log("SM %d blknum %d\n", sm, blknum.x);
	assert(sm <= blknum.x);
	assert(_vcast(trig[sm].from_device) != THREAD_WORK);

	_vcast(trig[sm].to_device) = THREAD_WORK;
	checkCudaErrors(hipMemcpyAsync(&d_trig[sm], &trig[sm], sizeof(trig_t), hipMemcpyHostToDevice, *backbone_stream));

	log("sm %d from_device %s to_device %s\n", sm,
        getFlagName(_vcast(trig[sm].from_device)), getFlagName(_vcast(trig[sm].to_device)));
}


/* Order the given sm to start working.
 */
void lkTriggerMultiple(trig_t *trig, trig_t *d_trig, dim3 blknum, hipStream_t *backbone_stream)
{
    log("blknum=%d\n", blknum.x);

    for(int i=0; i<blknum.x; i++)
    {
      log("Triggering SM #%d\n", i);
      _vcast(trig[i].to_device) = THREAD_WORK;
    }
    
    log("Transfering %d mailboxes to Device..\n", blknum.x);
    checkCudaErrors(hipMemcpyAsync(&d_trig[0], &trig[0], sizeof(trig_t) * blknum.x, hipMemcpyHostToDevice, *backbone_stream));
    
//     log("sm %d from_device %s to_device %s\n", sm,
//         getFlagName(_vcast(trig[sm].from_device)), getFlagName(_vcast(trig[sm].to_device)));
}

/* Busy wait until the given sm is working. Trigger to_device is restored to state "THREAD_NOP".
 * Formerly known as 'sm_wait'
 */
void lkWaitSM(trig_t *trig, trig_t *d_trig, int sm, dim3 blknum, hipStream_t *backbone_stream)
{
    log("SM #%d\n", sm);
    
    if(_vcast(trig[sm].to_device) != THREAD_WORK)
    {
      printf("SM #%d was not triggered! %d", sm, _vcast(trig[sm].to_device));
      return;
    }
    
    log("waiting for SM #%d to start working\n",sm);

	do {

//         checkCudaErrors(hipMemcpyAsync(trig, d_trig, sizeof(trig_t)*blknum.x,
//               hipMemcpyDeviceToHost, *backbone_stream));
		checkCudaErrors(hipMemcpyAsync(&trig[sm], &d_trig[sm], sizeof(trig_t), hipMemcpyDeviceToHost, *backbone_stream));
//         log("waiting for SM #%d to start working (to_device flag: %s, from_device flag: %s)\n",
//             sm, getFlagName(_vcast(trig[sm].to_device)), getFlagName(_vcast(trig[sm].from_device)));
        
	} while (_vcast(trig[sm].from_device) != THREAD_WORKING && _vcast(trig[sm].from_device) != THREAD_FINISHED);

    log("SM #%d is working: waiting for it to end\n", sm);
	do {

		checkCudaErrors(hipMemcpyAsync(&trig[sm], &d_trig[sm], sizeof(trig_t), hipMemcpyDeviceToHost, *backbone_stream));
// 		log("waiting for SM #%d to end working (to_device flag: %s, from_device flag: %s)\n",
//             sm, getFlagName(_vcast(trig[sm].to_device)), getFlagName(_vcast(trig[sm].from_device)));
	} while (_vcast(trig[sm].from_device) == THREAD_WORKING);
    
    log("SM #%d ended its work\n", sm);
}


/* Order to the kernel to exit and wait for its termination.
 * Formerly known as 'dispose'
 */
void lkDispose(trig_t *trig, trig_t *d_trig, dim3 blknum, hipStream_t *backbone_stream)
{
    int wg = blknum.x;
    log("Stop 'em!\n");

    for (int i = 0; i < wg; i++)
        _vcast(trig[i].to_device) = THREAD_EXIT;
    
    checkCudaErrors(hipMemcpyAsync(d_trig, trig, sizeof(trig_t) * wg, hipMemcpyHostToDevice, *backbone_stream));
    
    hipStreamSynchronize(*backbone_stream);

    checkCudaErrors(hipDeviceSynchronize());
    
    checkCudaErrors(hipGetLastError());

    checkCudaErrors(hipFree(d_trig));
    checkCudaErrors(hipHostFree(trig));
    
    checkCudaErrors(hipDeviceReset());
    log("Done.\n");
}

/* User-level API to embed LK into application */

void LKInit()
{
#if 0
  dim3 blknum = 1;
  dim3 blkdim = (1);
  int shmem = 0;
  char s[10000];
  long wait_total = 0, work_total = 0, assign_total = 0, retrieve_total = 0;
  
  verb("Warning: with VERBOSE flag on, time measures will be unreliable\n");

  /** BOOT (INIT) **/
  
  hipDeviceReset();
  
  log("LIGHTKERNEL START\n");

  int deviceCount;
  hipGetDeviceCount(&deviceCount);
  /* Get device properties */
  int device;
  for (device = 0; device < deviceCount; ++device)
  {
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, device);
    log("[boot] Device canMapHostMemory: %s.\n", deviceProp.canMapHostMemory ? "yes" : "no");
    log("[boot] Device %d has async engine count %d.\n", device, deviceProp.asyncEngineCount);
  }

  hipStream_t stream_kernel, backbone_stream;
  checkCudaErrors(hipStreamCreate(&stream_kernel));
  checkCudaErrors(hipStreamCreate(&backbone_stream));
  
  log("[boot] Number of Blocks: %d number of threads per block: %d, shared memory dim: %d\n", blknum.x, blkdim.x, shmem);

  int wg = blknum.x;

  struct timespec spec_start, spec_stop;
  
  trig_t *trig, *d_trig;
  data_t *data;
  int *lk_results;

  /** ALLOC (INIT) **/
  /* hipHostAlloc: shared between host and GPU */
  checkCudaErrors(hipHostAlloc((void **)&trig, wg * sizeof(trig_t), hipHostMallocDefault));
  checkCudaErrors(hipMalloc((void **)&d_trig, wg * sizeof(trig_t)));

  /* Call application-specific initialization of data
   * 'Big offload' is performed here */
  lkInitAppData(&data, wg);
  checkCudaErrors(hipHostAlloc((void **)&lk_results, wg * sizeof(int), hipHostMallocDefault));
  sprintf(s, "%s %ld", s, clock_getdiff_nsec(spec_start, spec_stop));
  verb("alloc(init) %lld\n", clock_getdiff_nsec(spec_start, spec_stop));

  /** LAUNCH (INIT) **/
  GETTIME_TIC;
  if(cudaMode)
    lkLaunch(lkUniformPollingCuda, trig, d_trig, data, lk_results, blkdim, blknum, shmem, &stream_kernel, &backbone_stream);
  else
    lkLaunch(lkUniformPollingNoCuda, trig, d_trig, data, lk_results, blkdim, blknum, shmem, &stream_kernel, &backbone_stream);
  
  GETTIME_TOC;
  sprintf(s, "%s %ld", s, clock_getdiff_nsec(spec_start, spec_stop));
  verb("launch(init) %lld\n", clock_getdiff_nsec(spec_start, spec_stop));
  //print_trigger("after init", trig);
#endif
}
